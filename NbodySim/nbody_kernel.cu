#include "hip/hip_runtime.h"
﻿// nbody_kernel.cu
#include "hip/hip_runtime.h"
#include ""
#include "nbody.cuh"
#include <cmath> // For sqrtf and fabsf


__global__ void updateParticles(Particle* particles, int numParticles, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        float2 force = { 0.0f, 0.0f };
        for (int j = 0; j < numParticles; ++j) {
            if (i != j) {
                float dx = particles[j].pos.x - particles[i].pos.x;
                float dy = particles[j].pos.y - particles[i].pos.y;
                float dist = sqrtf(dx * dx + dy * dy);
                float f = 0.001f / (dist * dist + 0.01f); // Simplified gravity with softening
                force.x += f * dx;
                force.y += f * dy;
            }
        }
        particles[i].vel.x += force.x * dt;
        particles[i].vel.y += force.y * dt;
        particles[i].pos.x += particles[i].vel.x * dt;
        particles[i].pos.y += particles[i].vel.y * dt;
    }
}

//// CUDA Kernel to compute forces
//__global__ void computeForces(Particle* particles, float* forcesX, float* forcesY, int numParticles) {
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if (i < numParticles) {
//        float fx = 0.0f, fy = 0.0f;
//        for (int j = 0; j < numParticles; ++j) {
//            if (i != j) {
//                float dx = particles[j].x - particles[i].x;
//                float dy = particles[j].y - particles[i].y;
//                float distSqr = dx * dx + dy * dy + SOFTENING;
//                float invDist = 1.0f / sqrtf(distSqr);
//                float invDistCube = invDist * invDist * invDist;
//                float f = G * particles[i].mass * particles[j].mass * invDistCube;
//                fx += f * dx;
//                fy += f * dy;
//            }
//        }
//        forcesX[i] = fx;
//        forcesY[i] = fy;
//    }
//}